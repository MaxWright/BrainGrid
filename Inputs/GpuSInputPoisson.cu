#include "hip/hip_runtime.h"
/*
 *      \file GpuSInputPoisson.cu
 *
 *      \author Fumitaka Kawasaki
 *
 *      \brief A class that performs stimulus input (implementation Poisson) on GPU.
 */

#include "hiprand/hiprand_kernel.h"
#include "GpuSInputPoisson.h"
#include <hip/hip_runtime_api.h>

//! Memory to save global state for hiprand.
hiprandState* devStates_d;

/*
 * constructor
 *
 * @param[in] psi       Pointer to the simulation information
 * @param[in] parms     TiXmlElement to examine.
 */
GpuSInputPoisson::GpuSInputPoisson(SimulationInfo* psi, TiXmlElement* parms) : SInputPoisson(psi, parms)
{
}

/*
 * destructor
 */
GpuSInputPoisson::~GpuSInputPoisson()
{
}

/*
 * Initialize data.
 *
 * @param[in] psi       Pointer to the simulation information.
 */
void GpuSInputPoisson::init(SimulationInfo* psi, ClusterInfo *pci)
{
    SInputPoisson::init(psi, pci);

    if (fSInput == false)
        return;

    // allocate GPU device memory and copy values
    allocDeviceValues(psi->model, psi, pci, nISIs);

    // CUDA parameters
    int neuron_count = pci->totalClusterNeurons;
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;

    // setup seeds
    setupSeeds <<< blocksPerGrid, threadsPerBlock >>> ( neuron_count, devStates_d, time(NULL) );
}

/*
 * Terminate process.
 *
 * @param[in] psi                Pointer to the simulation information.
 */
void GpuSInputPoisson::term(SimulationInfo* psi)
{
    SInputPoisson::term(psi);

    if (fSInput)
        deleteDeviceValues(psi->model, psi);
}

/*
 * Process input stimulus for each time step.
 * Apply inputs on summationPoint.
 *
 * @param[in] psi                Pointer to the simulation information.
 */
void GpuSInputPoisson::inputStimulus(const SimulationInfo* psi, const ClusterInfo* pci)
{
    if (fSInput == false)
        return;

    int neuron_count = pci->totalClusterNeurons;
    int synapse_count = pci->totalClusterNeurons;

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;

    // add input spikes to each synapse
    inputStimulusDevice <<< blocksPerGrid, threadsPerBlock >>> ( neuron_count, nISIs_d, masks_d, psi->deltaT, lambda, devStates_d, allSynapsesDevice );

    // advance synapses
    advanceSpikingSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( synapse_count, synapseIndexMapDevice, g_simulationStep, psi->deltaT, (AllSpikingSynapsesDeviceProperties*)allSynapsesDevice );

    advanceSpikingSynapsesEventQueueDevice <<< 1, 1 >>> ((AllSpikingSynapsesDeviceProperties*)allSynapsesDevice);

    // update summation point
    applyI2SummationMap <<< blocksPerGrid, threadsPerBlock >>> ( neuron_count, pci->pClusterSummationMap, allSynapsesDevice );
}

/*
 * Allocate GPU device memory and copy values
 *
 * @param[in] model      Pointer to the Neural Network Model object.
 * @param[in] psi        Pointer to the simulation information.
 * @param[in] nISIs      Pointer to the interval counter.
 */
void GpuSInputPoisson::allocDeviceValues(IModel* model, SimulationInfo* psi, ClusterInfo* pci, int *nISIs )
{
    int neuron_count = pci->totalClusterNeurons;
    BGSIZE nISIs_d_size = neuron_count * sizeof (int);   // size of shift values

    // Allocate GPU device memory
    checkCudaErrors( hipMalloc ( ( void ** ) &nISIs_d, nISIs_d_size ) );

    // Copy values into device memory
    checkCudaErrors( hipMemcpy ( nISIs_d, nISIs, nISIs_d_size, hipMemcpyHostToDevice ) );

    // create an input synapse layer
    m_synapses->allocSynapseDeviceStruct( (void **)&allSynapsesDevice, neuron_count, 1 ); 
    m_synapses->copySynapseHostToDevice( allSynapsesDevice, neuron_count, 1 );

    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;

    initSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( neuron_count, allSynapsesDevice, pci->pClusterSummationMap, psi->width, psi->deltaT, weight );

    // allocate memory for hiprand global state
    checkCudaErrors( hipMalloc ( &devStates_d, neuron_count * sizeof( hiprandState ) ) );

    // allocate memory for synapse index map and initialize it
    SynapseIndexMap synapseIndexMap;
    BGSIZE* incomingSynapseIndexMap = new BGSIZE[neuron_count];

    BGSIZE syn_i = 0;
    for (int i = 0; i < neuron_count; i++, syn_i++)
    {
        incomingSynapseIndexMap[i] = syn_i;
    }
    checkCudaErrors( hipMalloc( ( void ** ) &synapseIndexMap.incomingSynapseIndexMap, neuron_count * sizeof( BGSIZE ) ) );
    checkCudaErrors( hipMemcpy ( synapseIndexMap.incomingSynapseIndexMap, incomingSynapseIndexMap, neuron_count * sizeof( BGSIZE ), hipMemcpyHostToDevice ) ); 
    checkCudaErrors( hipMalloc( ( void ** ) &synapseIndexMapDevice, sizeof( SynapseIndexMap ) ) );
    checkCudaErrors( hipMemcpy ( synapseIndexMapDevice, &synapseIndexMap, sizeof( SynapseIndexMap ), hipMemcpyHostToDevice ) );

    delete[] incomingSynapseIndexMap;

    // allocate memory for masks for stimulus input and initialize it
    checkCudaErrors( hipMalloc ( &masks_d, neuron_count * sizeof( bool ) ) );
    checkCudaErrors( hipMemcpy ( masks_d, masks, neuron_count * sizeof( bool ), hipMemcpyHostToDevice ) ); 
}

/*
 * Dellocate GPU device memory
 *
 * @param[in] model      Pointer to the Neural Network Model object.
 * @param[in] psi        Pointer to the simulation information.
 */
void GpuSInputPoisson::deleteDeviceValues(IModel* model, SimulationInfo* psi )
{
    checkCudaErrors( hipFree( nISIs_d ) );
    checkCudaErrors( hipFree( devStates_d ) );
    checkCudaErrors( hipFree( masks_d ) );

    m_synapses->deleteSynapseDeviceStruct( allSynapsesDevice );

    // deallocate memory for synapse index map
    SynapseIndexMap synapseIndexMap;
    checkCudaErrors( hipMemcpy ( &synapseIndexMap, synapseIndexMapDevice, sizeof( SynapseIndexMap ), hipMemcpyDeviceToHost ) );
    checkCudaErrors( hipFree( synapseIndexMap.incomingSynapseIndexMap ) );
    checkCudaErrors( hipFree( synapseIndexMapDevice ) );
}

// CUDA code for -----------------------------------------------------------------------

/*
 * Device code for adding input values to the summation map.
 *
 * @param[in] nISIs_d            Pointer to the interval counter.
 * @param[in] masks_d            Pointer to the input stimulus masks.
 * @param[in] deltaT             Time step of the simulation in second.
 * @param[in] lambda             Iinverse firing rate.
 * @param[in] devStates_d        Curand global state
 * @param[in] allSynapsesDevice  Pointer to Synapse structures in device memory.
 */
__global__ void inputStimulusDevice( int n, int* nISIs_d, bool* masks_d, BGFLOAT deltaT, BGFLOAT lambda, hiprandState* devStates_d, AllDSSynapsesDeviceProperties* allSynapsesDevice )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= n )
        return;

    if (masks_d[idx] == false)
        return;

    BGSIZE iSyn = idx;

    int rnISIs = nISIs_d[idx];    // load the value to a register
    if (--rnISIs <= 0)
    {
        // add a spike
        int total_delay = allSynapsesDevice->total_delay[iSyn];
        allSynapsesDevice->preSpikeQueue->addAnEvent(iSyn, total_delay);

        // update interval counter (exponectially distribution ISIs, Poisson)
        hiprandState localState = devStates_d[idx];

        BGFLOAT isi = -lambda * log(hiprand_uniform( &localState ));
        // delete isi within refractoriness
        while (hiprand_uniform( &localState ) <= exp(-(isi*isi)/32))
            isi = -lambda * log(hiprand_uniform( &localState ));
        // convert isi from msec to steps
        rnISIs = static_cast<int>( (isi / 1000) / deltaT + 0.5 );
        devStates_d[idx] = localState;
    }
    nISIs_d[idx] = rnISIs;
}

/*
 * CUDA code for update summation point
 *
 * @param[in] n                  Number of neurons.
 * @param[in] summationPoint_d   SummationPoint
 * @param[in] allSynapsesDevice  Pointer to Synapse structures in device memory.
 */
__global__ void applyI2SummationMap( int n, BGFLOAT* summationPoint_d, AllDSSynapsesDeviceProperties* allSynapsesDevice ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= n )
            return;

    summationPoint_d[idx] += allSynapsesDevice->psr[idx];
}

/*
 * CUDA code for setup hiprand seed
 *
 * @param[in] n                  Number of neurons.
 * @param[in] devStates_d        Curand global state
 * @param[in] seed               Seed
 */
__global__ void setupSeeds( int n, hiprandState* devStates_d, unsigned long seed )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= n )
            return;

    hiprand_init( seed, idx, 0, &devStates_d[idx] );
} 
