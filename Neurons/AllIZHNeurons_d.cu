/*
 * AllIZHNeurons.cu
 *
 */

#include "AllSpikingSynapses.h"
#include "AllIZHNeurons.h"
#include "AllNeuronsDeviceFuncs.h"

#include <hip/hip_runtime_api.h>

/*
 *  Allocate GPU memories to store all neurons' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::allocNeuronDeviceStruct( void** allNeuronsDevice, SimulationInfo *sim_info ) {
	AllIZHNeuronsDeviceProperties allNeurons;

	allocDeviceStruct( allNeurons, sim_info );

        checkCudaErrors( hipMalloc( allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ) ) );
        checkCudaErrors( hipMemcpy ( *allNeuronsDevice, &allNeurons, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all neurons' states.
 *  (Helper function of allocNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the AllIZHNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::allocDeviceStruct( AllIZHNeuronsDeviceProperties &allNeurons, SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	AllIFNeurons::allocDeviceStruct( allNeurons, sim_info );
 
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Aconst, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Bconst, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Cconst, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Dconst, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.u, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.C3, count * sizeof( BGFLOAT ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::deleteNeuronDeviceStruct( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIZHNeuronsDeviceProperties allNeurons;

	checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allNeurons, sim_info );

	checkCudaErrors( hipFree( allNeuronsDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the AllIZHNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::deleteDeviceStruct( AllIZHNeuronsDeviceProperties& allNeurons, const SimulationInfo *sim_info ) {
	checkCudaErrors( hipFree( allNeurons.Aconst ) );
	checkCudaErrors( hipFree( allNeurons.Bconst ) );
	checkCudaErrors( hipFree( allNeurons.Cconst ) );
	checkCudaErrors( hipFree( allNeurons.Dconst ) );
	checkCudaErrors( hipFree( allNeurons.u ) );
	checkCudaErrors( hipFree( allNeurons.C3 ) );

	AllIFNeurons::deleteDeviceStruct( allNeurons, sim_info );
}

/*
 *  Copy all neurons' data from host to device.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::copyNeuronHostToDevice( void* allNeuronsDevice, const SimulationInfo *sim_info ) { 
	AllIZHNeuronsDeviceProperties allNeurons;

	checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allNeurons, sim_info );
}

/*
 *  Copy all neurons' data from host to device.
 *  (Helper function of copyNeuronHostToDevice)
 *
 *  @param  allNeurons         Reference to the AllIZHNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::copyHostToDevice( AllIZHNeuronsDeviceProperties& allNeurons, const SimulationInfo *sim_info ) { 
	int count = sim_info->totalNeurons;

	AllIFNeurons::copyHostToDevice( allNeurons, sim_info );

	checkCudaErrors( hipMemcpy ( allNeurons.Aconst, Aconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Bconst, Bconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Cconst, Cconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Dconst, Dconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.u, u, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.C3, C3, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all neurons' data from device to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceToHost( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIZHNeuronsDeviceProperties allNeurons;

	checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allNeurons, sim_info );
}

/*
 *  Copy all neurons' data from device to host.
 *  (Helper function of copyNeuronDeviceToHost)
 *
 *  @param  allNeurons         Reference to the AllIZHNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::copyDeviceToHost( AllIZHNeuronsDeviceProperties& allNeurons, const SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	AllIFNeurons::copyDeviceToHost( allNeurons, sim_info );

	checkCudaErrors( hipMemcpy ( Aconst, allNeurons.Aconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Bconst, allNeurons.Bconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Cconst, allNeurons.Cconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Dconst, allNeurons.Dconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( u, allNeurons.u, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( C3, allNeurons.C3, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/*
 *  Copy spike history data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
        AllIZHNeuronsDeviceProperties allNeurons;
        checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allNeurons, sim_info );
}

/*
 *  Copy spike counts data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceSpikeCountsToHost( void* allNeuronsDevice, const SimulationInfo *sim_info )
{
        AllIZHNeuronsDeviceProperties allNeurons;
        checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allNeurons, sim_info );
}

/*
 *  Clear the spike counts out of all neurons.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::clearNeuronSpikeCounts( void* allNeuronsDevice, const SimulationInfo *sim_info )
{
        AllIZHNeuronsDeviceProperties allNeurons;
        checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::clearDeviceSpikeCounts( allNeurons, sim_info );
}

/*
 *  Notify outgoing synapses if neuron has fired.
 *
 *  @param  sim_info    SimulationInfo class to read information from.
 */
void AllIZHNeurons::advanceNeurons( IAllSynapses &synapses, void* allNeuronsDevice, void* allSynapsesDevice, const SimulationInfo *sim_info, float* randNoise, SynapseIndexMap* synapseIndexMapDevice)
{
    int neuron_count = sim_info->totalNeurons;
    int maxSpikes = (int)((sim_info->epochDuration * sim_info->maxFiringRate));

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance neurons ------------->
    advanceIZHNeuronsDevice <<< blocksPerGrid, threadsPerBlock >>> ( neuron_count, sim_info->maxSynapsesPerNeuron, maxSpikes, sim_info->deltaT, g_simulationStep, randNoise, (AllIZHNeuronsDeviceProperties *)allNeuronsDevice, (AllSpikingSynapsesDeviceProperties*)allSynapsesDevice, synapseIndexMapDevice, m_fAllowBackPropagation );
}

