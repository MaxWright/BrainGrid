/*
 * AllSpikingNeurons_d.cu
 *
 */

#include "AllSpikingNeurons.h"
#include "AllSpikingSynapses.h"
#include <hip/hip_runtime_api.h>

/*
 *  Copy spike history data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllSpikingNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllSpikingNeurons::copyDeviceSpikeHistoryToHost( AllSpikingNeuronsDeviceProperties& allNeurons, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) 
{
        int numNeurons = clr_info->totalClusterNeurons;
        uint64_t* pSpikeHistory[numNeurons];
        checkCudaErrors( hipMemcpy ( pSpikeHistory, allNeurons.spike_history, numNeurons * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );

        int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
        for (int i = 0; i < numNeurons; i++) {
                checkCudaErrors( hipMemcpy ( spike_history[i], pSpikeHistory[i],
                        max_spikes * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        }
}

/*
 *  Copy spike counts data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllSpikingNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllSpikingNeurons::copyDeviceSpikeCountsToHost( AllSpikingNeuronsDeviceProperties& allNeurons, const ClusterInfo *clr_info ) 
{
        int numNeurons = clr_info->totalClusterNeurons;

        checkCudaErrors( hipMemcpy ( spikeCount, allNeurons.spikeCount, numNeurons * sizeof( int ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( spikeCountOffset, allNeurons.spikeCountOffset, numNeurons * sizeof( int ), hipMemcpyDeviceToHost ) );
}

/*
 *  Clear the spike counts out of all neurons in device memory.
 *  (helper function of clearNeuronSpikeCounts)
 *
 *  @param  allNeurons         Reference to the allNeurons struct.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllSpikingNeurons::clearDeviceSpikeCounts( AllSpikingNeuronsDeviceProperties& allNeurons, const ClusterInfo *clr_info ) 
{
        int numNeurons = clr_info->totalClusterNeurons;

        checkCudaErrors( hipMemset( allNeurons.spikeCount, 0, numNeurons * sizeof( int ) ) );
        checkCudaErrors( hipMemcpy ( allNeurons.spikeCountOffset, spikeCountOffset, numNeurons * sizeof( int ), hipMemcpyHostToDevice ) );
}

/*
 *  Set some parameters used for advanceNeuronsDevice.
 *  Currently we set the two member variables: m_fpPreSpikeHit_h and m_fpPostSpikeHit_h.
 *  These are function pointers for PreSpikeHit and PostSpikeHit device functions
 *  respectively, and these functions are called from advanceNeuronsDevice device
 *  function. We use this scheme because we cannot not use virtual function (Polymorphism)
 *  in device functions.
 *
 *  @param  synapses               Reference to the allSynapses struct on host memory.
 */
void AllSpikingNeurons::setAdvanceNeuronsDeviceParams(IAllSynapses &synapses)
{
    AllSpikingSynapses &spSynapses = dynamic_cast<AllSpikingSynapses&>(synapses);
    m_fAllowBackPropagation = spSynapses.allowBackPropagation();
}
