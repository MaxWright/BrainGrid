/*
 * AllIFNeurons_d.cu
 *
 */

#include "AllIFNeurons.h"
#include <hip/hip_runtime_api.h>

/*
 *  Allocate GPU memories to store all neurons' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::allocNeuronDeviceStruct( void** allNeuronsDevice, SimulationInfo *sim_info, ClusterInfo *clr_info ) {
	AllIFNeuronsDeviceProperties allNeurons;

	allocDeviceStruct( allNeurons, sim_info, clr_info );

        checkCudaErrors( hipMalloc( allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ) ) );
        checkCudaErrors( hipMemcpy ( *allNeuronsDevice, &allNeurons, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all neurons' states.
 *  (Helper function of allocNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the AllIFNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::allocDeviceStruct( AllIFNeuronsDeviceProperties &allNeurons, SimulationInfo *sim_info, ClusterInfo *clr_info ) {
	int count = clr_info->totalClusterNeurons;
	int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
 
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.C1, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.C2, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Cm, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.I0, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Iinject, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Inoise, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Isyn, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Rm, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Tau, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Trefract, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Vinit, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Vm, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Vreset, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Vrest, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Vthresh, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.hasFired, count * sizeof( bool ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.nStepsInRefr, count * sizeof( int ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.spikeCount, count * sizeof( int ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.spikeCountOffset, count * sizeof( int ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.summation_map, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.spike_history, count * sizeof( uint64_t* ) ) );
	
	uint64_t* pSpikeHistory[count];
	for (int i = 0; i < count; i++) {
		checkCudaErrors( hipMalloc( ( void ** ) &pSpikeHistory[i], max_spikes * sizeof( uint64_t ) ) );
	}
	checkCudaErrors( hipMemcpy ( allNeurons.spike_history, pSpikeHistory,
		count * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );

	// get device summation point address and set it to sim info
	clr_info->pClusterSummationMap = allNeurons.summation_map;
}

/*
 *  Delete GPU memories.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::deleteNeuronDeviceStruct( void* allNeuronsDevice, const ClusterInfo *clr_info ) {
	AllIFNeuronsDeviceProperties allNeurons;

	checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allNeurons, clr_info );

	checkCudaErrors( hipFree( allNeuronsDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the AllIFNeuronsDeviceProperties struct.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::deleteDeviceStruct( AllIFNeuronsDeviceProperties& allNeurons, const ClusterInfo *clr_info ) {
	int count = clr_info->totalClusterNeurons;

	uint64_t* pSpikeHistory[count];
	checkCudaErrors( hipMemcpy ( pSpikeHistory, allNeurons.spike_history,
		count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	for (int i = 0; i < count; i++) {
		checkCudaErrors( hipFree( pSpikeHistory[i] ) );
	}

	checkCudaErrors( hipFree( allNeurons.C1 ) );
	checkCudaErrors( hipFree( allNeurons.C2 ) );
	checkCudaErrors( hipFree( allNeurons.Cm ) );
	checkCudaErrors( hipFree( allNeurons.I0 ) );
	checkCudaErrors( hipFree( allNeurons.Iinject ) );
	checkCudaErrors( hipFree( allNeurons.Inoise ) );
	checkCudaErrors( hipFree( allNeurons.Isyn ) );
	checkCudaErrors( hipFree( allNeurons.Rm ) );
	checkCudaErrors( hipFree( allNeurons.Tau ) );
	checkCudaErrors( hipFree( allNeurons.Trefract ) );
	checkCudaErrors( hipFree( allNeurons.Vinit ) );
	checkCudaErrors( hipFree( allNeurons.Vm ) );
	checkCudaErrors( hipFree( allNeurons.Vreset ) );
	checkCudaErrors( hipFree( allNeurons.Vrest ) );
	checkCudaErrors( hipFree( allNeurons.Vthresh ) );
	checkCudaErrors( hipFree( allNeurons.hasFired ) );
	checkCudaErrors( hipFree( allNeurons.nStepsInRefr ) );
	checkCudaErrors( hipFree( allNeurons.spikeCount ) );
	checkCudaErrors( hipFree( allNeurons.spikeCountOffset ) );
	checkCudaErrors( hipFree( allNeurons.summation_map ) );
	checkCudaErrors( hipFree( allNeurons.spike_history ) );
}

/*
 *  Copy all neurons' data from host to device.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::copyNeuronHostToDevice( void* allNeuronsDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) { 
	AllIFNeuronsDeviceProperties allNeurons;

	checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allNeurons, sim_info, clr_info );
}

/*
 *  Copy all neurons' data from host to device.
 *  (Helper function of copyNeuronHostToDevice)
 *
 *  @param  allNeurons         Reference to the AllIFNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::copyHostToDevice( AllIFNeuronsDeviceProperties& allNeurons, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) { 
	int count = clr_info->totalClusterNeurons;

	checkCudaErrors( hipMemcpy ( allNeurons.C1, C1, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.C2, C2, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Cm, Cm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.I0, I0, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Iinject, Iinject, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Inoise, Inoise, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Isyn, Isyn, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Rm, Rm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Tau, Tau, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Trefract, Trefract, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Vinit, Vinit, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Vm, Vm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Vreset, Vreset, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Vrest, Vrest, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Vthresh, Vthresh, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.hasFired, hasFired, count * sizeof( bool ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.nStepsInRefr, nStepsInRefr, count * sizeof( int ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.spikeCount, spikeCount, count * sizeof( int ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.spikeCountOffset, spikeCountOffset, count * sizeof( int ), hipMemcpyHostToDevice ) );

        int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
        uint64_t* pSpikeHistory[count];
        checkCudaErrors( hipMemcpy ( pSpikeHistory, allNeurons.spike_history, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
        for (int i = 0; i < count; i++) {
                checkCudaErrors( hipMemcpy ( pSpikeHistory[i], spike_history[i], max_spikes * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        }
}

/*
 *  Copy all neurons' data from device to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::copyNeuronDeviceToHost( void* allNeuronsDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
	AllIFNeuronsDeviceProperties allNeurons;

	checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allNeurons, sim_info, clr_info );
}

/*
 *  Copy all neurons' data from device to host.
 *  (Helper function of copyNeuronDeviceToHost)
 *
 *  @param  allNeurons         Reference to the AllIFNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::copyDeviceToHost( AllIFNeuronsDeviceProperties& allNeurons, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
	int count = clr_info->totalClusterNeurons;

	checkCudaErrors( hipMemcpy ( C1, allNeurons.C1, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( C2, allNeurons.C2, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Cm, allNeurons.C1, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( I0, allNeurons.I0, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Iinject, allNeurons.Iinject, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Inoise, allNeurons.Inoise, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Isyn, allNeurons.Isyn, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Rm, allNeurons.Rm, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Tau, allNeurons.Tau, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Trefract, allNeurons.Trefract, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Vinit, allNeurons.Vinit, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Vm, allNeurons.Vm, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Vreset, allNeurons.Vreset, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Vrest, allNeurons.Vrest, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Vthresh, allNeurons.Vthresh, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( hasFired, allNeurons.hasFired, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( nStepsInRefr, allNeurons.nStepsInRefr, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( spikeCount, allNeurons.spikeCount, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( spikeCountOffset, allNeurons.spikeCountOffset, count * sizeof( int ), hipMemcpyDeviceToHost ) );

        int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
        uint64_t* pSpikeHistory[count];
        checkCudaErrors( hipMemcpy ( pSpikeHistory, allNeurons.spike_history, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
        for (int i = 0; i < count; i++) {
                checkCudaErrors( hipMemcpy ( spike_history[i], pSpikeHistory[i], max_spikes * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        }
}

/*
 *  Copy spike history data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allNeuronsDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) 
{        
        AllIFNeuronsDeviceProperties allNeurons;
        checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );        
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allNeurons, sim_info, clr_info );
}

/*
 *  Copy spike counts data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::copyNeuronDeviceSpikeCountsToHost( void* allNeuronsDevice, const ClusterInfo *clr_info )
{
        AllIFNeuronsDeviceProperties allNeurons;
        checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allNeurons, clr_info );
}

/*
 *  Clear the spike counts out of all neurons.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::clearNeuronSpikeCounts( void* allNeuronsDevice, const ClusterInfo *clr_info )
{
        AllIFNeuronsDeviceProperties allNeurons;
        checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::clearDeviceSpikeCounts( allNeurons, clr_info );
}

/*
 *  Update the state of all neurons for a time step
 *  Notify outgoing synapses if neuron has fired.
 *
 *  @param  synapses               Reference to the allSynapses struct on host memory.
 *  @param  allNeuronsDevice       Reference to the AllIFNeuronsDeviceProperties struct 
 *                                 on device memory.
 *  @param  allSynapsesDevice      Reference to the allSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  sim_info               SimulationInfo to refer from.
 *  @param  randNoise              Reference to the random noise array.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 *  @param  clr_info               ClusterInfo to refer from.
 */
void AllIFNeurons::advanceNeurons( IAllSynapses &synapses, void* allNeuronsDevice, void* allSynapsesDevice, const SimulationInfo *sim_info, float* randNoise, SynapseIndexMap* synapseIndexMapDevice, const ClusterInfo *clr_info )
{
}
