/*
 * AllSTDPSynapses_d.cu
 *
 */

#include "AllSTDPSynapses.h"
#include "AllSpikingSynapses.h"
#include "GPUSpikingModel.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info ) {
	allocSynapseDeviceStruct( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     Reference to the AllSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) {
	AllSTDPSynapsesDeviceProperties allSynapses;

	allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

	HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllSTDPSynapsesDeviceProperties ) ) );
	HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice     Reference to the AllSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::allocDeviceStruct( AllSTDPSynapsesDeviceProperties &allSynapses, int num_neurons, int maxSynapsesPerNeuron ) {
        AllSpikingSynapses::allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.total_delayPost, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.tauspost, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.tauspre, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.taupos, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.tauneg, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.STDPgap, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.Wex, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.Aneg, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.Apos, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.mupos, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.muneg, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.useFroemkeDanSTDP, max_total_synapses * sizeof( bool ) ) );

        // create a EventQueue objet in device memory and set the pointer to postSpikeQueue.
        EventQueue **pEventQueue; // temporary buffer to save pointer to EventQueue object.

        // allocate device memory for the buffer.
        HANDLE_ERROR( hipMalloc( ( void ** ) &pEventQueue, sizeof( EventQueue * ) ) );

        // create a EventQueue object in device memory.
        allocEventQueueDevice <<< 1, 1 >>> ( max_total_synapses, pEventQueue );

        // save the pointer of the object.
        HANDLE_ERROR( hipMemcpy ( &allSynapses.postSpikeQueue, pEventQueue, sizeof( EventQueue * ), hipMemcpyDeviceToHost ) );

        // free device memory for the buffer.
        HANDLE_ERROR( hipFree( pEventQueue ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  Reference to the AllSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSTDPSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
	AllSTDPSynapsesDeviceProperties allSynapses;

	HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allSynapses );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the AllSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSTDPSynapses::deleteDeviceStruct( AllSTDPSynapsesDeviceProperties& allSynapses ) {
        HANDLE_ERROR( hipFree( allSynapses.total_delayPost ) );
        HANDLE_ERROR( hipFree( allSynapses.tauspost ) );
        HANDLE_ERROR( hipFree( allSynapses.tauspre ) );
        HANDLE_ERROR( hipFree( allSynapses.taupos ) );
        HANDLE_ERROR( hipFree( allSynapses.tauneg ) );
        HANDLE_ERROR( hipFree( allSynapses.STDPgap ) );
        HANDLE_ERROR( hipFree( allSynapses.Wex ) );
        HANDLE_ERROR( hipFree( allSynapses.Aneg ) );
        HANDLE_ERROR( hipFree( allSynapses.Apos ) );
        HANDLE_ERROR( hipFree( allSynapses.mupos ) );
        HANDLE_ERROR( hipFree( allSynapses.muneg ) );
        HANDLE_ERROR( hipFree( allSynapses.useFroemkeDanSTDP ) );

        // delete EventQueue object in device memory.
        deleteEventQueueDevice <<< 1, 1 >>> ( allSynapses.postSpikeQueue );

        AllSpikingSynapses::deleteDeviceStruct( allSynapses );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     Reference to the AllSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info ) { // copy everything necessary
	copySynapseHostToDevice( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     Reference to the AllSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
	AllSTDPSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice     Reference to the AllSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::copyHostToDevice( void* allSynapsesDevice, AllSTDPSynapsesDeviceProperties& allSynapses, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        AllSpikingSynapses::copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;
        
        HANDLE_ERROR( hipMemcpy ( allSynapses.total_delayPost, total_delayPost,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.tauspost, tauspost,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.tauspre, tauspre,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.taupos, taupos,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.tauneg, tauneg,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.STDPgap, STDPgap,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.Wex, Wex,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.Aneg, Aneg,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.Apos, Apos,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.mupos, mupos,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.muneg, muneg,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.useFroemkeDanSTDP, useFroemkeDanSTDP,
                max_total_synapses * sizeof( bool ), hipMemcpyHostToDevice ) ); 

        // deep copy postSpikeQueue from host to device
        BGQUEUE_ELEMENT* pQueueBuffer; // temporary buffer to save event queue.

        // allocate device memory for the buffer.
        HANDLE_ERROR( hipMalloc( ( void ** ) &pQueueBuffer, postSpikeQueue->m_nMaxEvent * sizeof( BGQUEUE_ELEMENT ) ) );

        // copy event queue data from host to the buffer.
        HANDLE_ERROR( hipMemcpy ( pQueueBuffer, postSpikeQueue->m_queueEvent, postSpikeQueue->m_nMaxEvent * sizeof( BGQUEUE_ELEMENT ), hipMemcpyHostToDevice ) );

        // copy event queue data from the buffer to the device.
        copyEventQueueDevice <<< 1, 1 >>> (allSynapses.postSpikeQueue, postSpikeQueue->m_nMaxEvent, postSpikeQueue->m_idxQueue, pQueueBuffer);

        // free device memory for the buffer.
        HANDLE_ERROR( hipFree( pQueueBuffer ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  Reference to the AllSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSTDPSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info ) {
	// copy everything necessary
	AllSTDPSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyDeviceToHost( allSynapses, sim_info );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice     Reference to the AllSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::copyDeviceToHost( AllSTDPSynapsesDeviceProperties& allSynapses, const SimulationInfo *sim_info ) {
        AllSpikingSynapses::copyDeviceToHost( allSynapses, sim_info ) ;

	int num_neurons = sim_info->totalNeurons;
	BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMemcpy ( tauspost, allSynapses.tauspost,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauspre, allSynapses.tauspre,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( taupos, allSynapses.taupos,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauneg, allSynapses.tauneg,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( STDPgap, allSynapses.STDPgap,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( Wex, allSynapses.Wex,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( Aneg, allSynapses.Aneg,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( Apos, allSynapses.Apos,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( mupos, allSynapses.mupos,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( muneg, allSynapses.muneg,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( useFroemkeDanSTDP, allSynapses.useFroemkeDanSTDP,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );

        // deep copy postSpikeQueue from device to host.
        BGQUEUE_ELEMENT* pQueueBuffer; // temporary buffer to save event queue.
        EventQueue* pDstEventQueue;    // temporary buffer to save EventQueue object.

        // allocate device memories for buffers.
        HANDLE_ERROR( hipMalloc( ( void ** ) &pQueueBuffer, postSpikeQueue->m_nMaxEvent * sizeof( BGQUEUE_ELEMENT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &pDstEventQueue, sizeof( EventQueue ) ) );

        // copy event queue data from device to the buffers.
        copyEventQueueDevice <<< 1, 1 >>> (allSynapses.postSpikeQueue, pQueueBuffer, pDstEventQueue);

        // copy data in the buffers to the event queue in host memory.
        HANDLE_ERROR( hipMemcpy ( postSpikeQueue->m_queueEvent, pQueueBuffer, postSpikeQueue->m_nMaxEvent * sizeof( BGQUEUE_ELEMENT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &postSpikeQueue->m_nMaxEvent, &pDstEventQueue->m_nMaxEvent, sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &postSpikeQueue->m_idxQueue, &pDstEventQueue->m_idxQueue, sizeof( uint32_t ), hipMemcpyDeviceToHost ) );

        // free device memories for buffers.
        HANDLE_ERROR( hipFree( pQueueBuffer ) );
        HANDLE_ERROR( hipFree( pDstEventQueue ) );
}

/*
 *  Advance all the Synapses in the simulation.
 *  Update the state of all synapses for a time step.
 *
 *  @param  allSynapsesDevice      Reference to the AllSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  allNeuronsDevice       Reference to the allNeurons struct on device memory.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 *  @param  sim_info               SimulationInfo class to read information from.
 */
void AllSTDPSynapses::advanceSynapses(void* allSynapsesDevice, void* allNeuronsDevice, void* synapseIndexMapDevice, const SimulationInfo *sim_info)
{
    int max_spikes = (int) ((sim_info->epochDuration * sim_info->maxFiringRate));

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( total_synapse_counts + threadsPerBlock - 1 ) / threadsPerBlock;
    // Advance synapses ------------->
    advanceSTDPSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( total_synapse_counts, (SynapseIndexMap*)synapseIndexMapDevice, g_simulationStep, sim_info->deltaT, (AllSTDPSynapsesDeviceProperties*)allSynapsesDevice, (AllSpikingNeuronsDeviceProperties*)allNeuronsDevice, max_spikes, sim_info->width );

    advanceSTDPSynapsesEventQueueDevice <<< 1, 1 >>> ( (AllSTDPSynapsesDeviceProperties*)allSynapsesDevice );
}

/**     
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllSTDPSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllSTDPSynapses;

    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses)) );
}
