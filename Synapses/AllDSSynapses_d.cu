/*
 * AllDSSynapses_d.cu
 *
 */

#include "AllDSSynapses.h"
#include "GPUSpikingModel.h"
#include "AllSynapsesDeviceFuncs.h"
#include <hip/hip_runtime_api.h>

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllDSSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllDSSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
	allocSynapseDeviceStruct( allSynapsesDevice, clr_info->totalClusterNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     Reference to the AllDSSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDSSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) {
	AllDSSynapsesDeviceProperties allSynapses;

	allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

	checkCudaErrors( hipMalloc( allSynapsesDevice, sizeof( AllDSSynapsesDeviceProperties ) ) );
	checkCudaErrors( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllDSSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice     Reference to the AllDSSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDSSynapses::allocDeviceStruct( AllDSSynapsesDeviceProperties &allSynapses, int num_neurons, int maxSynapsesPerNeuron ) {
        AllSpikingSynapses::allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.lastSpike, max_total_synapses * sizeof( uint64_t ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.r, max_total_synapses * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.u, max_total_synapses * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.D, max_total_synapses * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.U, max_total_synapses * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.F, max_total_synapses * sizeof( BGFLOAT ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  Reference to the AllDSSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDSSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
	AllDSSynapsesDeviceProperties allSynapses;

	checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allSynapses );

	checkCudaErrors( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the AllDSSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllDSSynapses::deleteDeviceStruct( AllDSSynapsesDeviceProperties& allSynapses ) {
        checkCudaErrors( hipFree( allSynapses.lastSpike ) );
	checkCudaErrors( hipFree( allSynapses.r ) );
	checkCudaErrors( hipFree( allSynapses.u ) );
	checkCudaErrors( hipFree( allSynapses.D ) );
	checkCudaErrors( hipFree( allSynapses.U ) );
	checkCudaErrors( hipFree( allSynapses.F ) );

        AllSpikingSynapses::deleteDeviceStruct( allSynapses );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  Reference to the AllDSSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllDSSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) { // copy everything necessary
	copySynapseHostToDevice( allSynapsesDevice, clr_info->totalClusterNeurons, sim_info->maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     Reference to the AllDSSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDSSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
	AllDSSynapsesDeviceProperties allSynapses;

        checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice     Reference to the AllDSSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDSSynapses::copyHostToDevice( void* allSynapsesDevice, AllDSSynapsesDeviceProperties& allSynapses, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        AllSpikingSynapses::copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        checkCudaErrors( hipMemcpy ( allSynapses.lastSpike, lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.r, r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.u, u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.D, D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.U, U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.F, F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  Reference to the AllDSSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllDSSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
	// copy everything necessary
	AllDSSynapsesDeviceProperties allSynapses;

        checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyDeviceToHost( allSynapses, sim_info, clr_info );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice     Reference to the AllDSSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllDSSynapses::copyDeviceToHost( AllDSSynapsesDeviceProperties& allSynapses, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
        AllSpikingSynapses::copyDeviceToHost( allSynapses, sim_info, clr_info ) ;

	int num_neurons = clr_info->totalClusterNeurons;
	BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        checkCudaErrors( hipMemcpy ( lastSpike, allSynapses.lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( r, allSynapses.r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( u, allSynapses.u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( D, allSynapses.D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( U, allSynapses.U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( F, allSynapses.F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/**     
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllDSSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllDSSynapses;

    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses)) );
}

