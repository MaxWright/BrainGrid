/*
 * AllSpikingSynapses.cu
 *
 */

#include "AllSpikingSynapses.h"
#include "AllSynapsesDeviceFuncs.h"
#include <hip/hip_runtime_api.h>

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info ) {
        allocSynapseDeviceStruct( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) {
        AllSpikingSynapsesDeviceProperties allSynapses;

        allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

        checkCudaErrors( hipMalloc( allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ) ) );
        checkCudaErrors( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::allocDeviceStruct( AllSpikingSynapsesDeviceProperties &allSynapses, int num_neurons, int maxSynapsesPerNeuron ) {
        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.sourceNeuronIndex, max_total_synapses * sizeof( int ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.destNeuronIndex, max_total_synapses * sizeof( int ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.W, max_total_synapses * sizeof( BGFLOAT ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.type, max_total_synapses * sizeof( synapseType ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.psr, max_total_synapses * sizeof( BGFLOAT ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.in_use, max_total_synapses * sizeof( bool ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.synapse_counts, num_neurons * sizeof( BGSIZE ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.decay, max_total_synapses * sizeof( BGFLOAT ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.tau, max_total_synapses * sizeof( BGFLOAT ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.total_delay, max_total_synapses * sizeof( int ) ) );

        // create a EventQueue objet in device memory and set the pointer to preSpikeQueue.
        EventQueue **pEventQueue; // temporary buffer to save pointer to EventQueue object.

        // allocate device memory for the buffer.
        checkCudaErrors( hipMalloc( ( void ** ) &pEventQueue, sizeof( EventQueue * ) ) );

        // create a EventQueue object in device memory.
        allocEventQueueDevice <<< 1, 1 >>> ( max_total_synapses, pEventQueue );

        // save the pointer of the object.
        checkCudaErrors( hipMemcpy ( &allSynapses.preSpikeQueue, pEventQueue, sizeof( EventQueue * ), hipMemcpyDeviceToHost ) );

        // free device memory for the buffer.
        checkCudaErrors( hipFree( pEventQueue ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
        AllSpikingSynapsesDeviceProperties allSynapses;

        checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        deleteDeviceStruct( allSynapses );

        checkCudaErrors( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::deleteDeviceStruct( AllSpikingSynapsesDeviceProperties& allSynapses ) {
        checkCudaErrors( hipFree( allSynapses.sourceNeuronIndex ) );
        checkCudaErrors( hipFree( allSynapses.destNeuronIndex ) );
        checkCudaErrors( hipFree( allSynapses.W ) );
        checkCudaErrors( hipFree( allSynapses.type ) );
        checkCudaErrors( hipFree( allSynapses.psr ) );
        checkCudaErrors( hipFree( allSynapses.in_use ) );
        checkCudaErrors( hipFree( allSynapses.synapse_counts ) );
        checkCudaErrors( hipFree( allSynapses.decay ) );
        checkCudaErrors( hipFree( allSynapses.tau ) );
        checkCudaErrors( hipFree( allSynapses.total_delay ) );

        // delete EventQueue object in device memory.
        deleteEventQueueDevice <<< 1, 1 >>> ( allSynapses.preSpikeQueue );

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.count_neurons = 0;
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info ) { // copy everything necessary
        copySynapseHostToDevice( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
        AllSpikingSynapsesDeviceProperties allSynapses;

        checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copyHostToDevice( void* allSynapsesDevice, AllSpikingSynapsesDeviceProperties& allSynapses, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        allSynapses.maxSynapsesPerNeuron = maxSynapsesPerNeuron;
        allSynapses.total_synapse_counts = total_synapse_counts;
        allSynapses.count_neurons = count_neurons;
        checkCudaErrors( hipMemcpy ( allSynapsesDevice, &allSynapses, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        allSynapses.count_neurons = 0;

        checkCudaErrors( hipMemcpy ( allSynapses.sourceNeuronIndex, sourceNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.destNeuronIndex, destNeuronIndex,
                max_total_synapses * sizeof( int ),  hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.W, W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.type, type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.psr, psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.in_use, in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.synapse_counts, synapse_counts,
                        num_neurons * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.decay, decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.tau, tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.total_delay, total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );

        // deep copy preSpikeQueue from host to device
        BGQUEUE_ELEMENT* pQueueBuffer; // temporary buffer to save event queue.

        // allocate device memory for the buffer.
        checkCudaErrors( hipMalloc( ( void ** ) &pQueueBuffer, preSpikeQueue->m_nMaxEvent * sizeof( BGQUEUE_ELEMENT ) ) );

        // copy event queue data from host to the buffer.
        checkCudaErrors( hipMemcpy ( pQueueBuffer, preSpikeQueue->m_queueEvent, preSpikeQueue->m_nMaxEvent * sizeof( BGQUEUE_ELEMENT ), hipMemcpyHostToDevice ) );

        // copy event queue data from the buffer to the device.
        copyEventQueueDevice <<< 1, 1 >>> (allSynapses.preSpikeQueue, preSpikeQueue->m_nMaxEvent, preSpikeQueue->m_idxQueue, pQueueBuffer);

        // free device memory for the buffer.
        checkCudaErrors( hipFree( pQueueBuffer ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info ) {
        // copy everything necessary
        AllSpikingSynapsesDeviceProperties allSynapses;

        checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        copyDeviceToHost( allSynapses, sim_info );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copyDeviceToHost( AllSpikingSynapsesDeviceProperties& allSynapses, const SimulationInfo *sim_info ) {
        int num_neurons = sim_info->totalNeurons;
        BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        checkCudaErrors( hipMemcpy ( synapse_counts, allSynapses.synapse_counts,
                num_neurons * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        maxSynapsesPerNeuron = allSynapses.maxSynapsesPerNeuron;
        total_synapse_counts = allSynapses.total_synapse_counts;
        count_neurons = allSynapses.count_neurons;

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        allSynapses.count_neurons = 0;

        checkCudaErrors( hipMemcpy ( sourceNeuronIndex, allSynapses.sourceNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( destNeuronIndex, allSynapses.destNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( W, allSynapses.W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( type, allSynapses.type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( psr, allSynapses.psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( in_use, allSynapses.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( decay, allSynapses.decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( tau, allSynapses.tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( total_delay, allSynapses.total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );

        // deep copy preSpikeQueue from device to host.
        BGQUEUE_ELEMENT* pQueueBuffer; // temporary buffer to save event queue.
        EventQueue* pDstEventQueue;    // temporary buffer to save EventQueue object.

        // allocate device memories for buffers.
        checkCudaErrors( hipMalloc( ( void ** ) &pQueueBuffer, preSpikeQueue->m_nMaxEvent * sizeof( BGQUEUE_ELEMENT ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &pDstEventQueue, sizeof( EventQueue ) ) );

        // copy event queue data from device to the buffers.
        copyEventQueueDevice <<< 1, 1 >>> (allSynapses.preSpikeQueue, pQueueBuffer, pDstEventQueue);

        // copy data in the buffers to the event queue in host memory.
        checkCudaErrors( hipMemcpy ( preSpikeQueue->m_queueEvent, pQueueBuffer, preSpikeQueue->m_nMaxEvent * sizeof( BGQUEUE_ELEMENT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( &preSpikeQueue->m_nMaxEvent, &pDstEventQueue->m_nMaxEvent, sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( &preSpikeQueue->m_idxQueue, &pDstEventQueue->m_idxQueue, sizeof( uint32_t ), hipMemcpyDeviceToHost ) );

        // free device memories for buffers.
        checkCudaErrors( hipFree( pQueueBuffer ) );
        checkCudaErrors( hipFree( pDstEventQueue ) );
}

/*
 *  Get synapse_counts in AllSynapses struct on device memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::copyDeviceSynapseCountsToHost(void* allSynapsesDevice, const SimulationInfo *sim_info)
{
        AllSpikingSynapsesDeviceProperties allSynapses;
        int neuron_count = sim_info->totalNeurons;

        checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( synapse_counts, allSynapses.synapse_counts, neuron_count * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.count_neurons = 0;
}

/* 
 *  Get summationCoord and in_use in AllSynapses struct on device memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::copyDeviceSynapseSumIdxToHost(void* allSynapsesDevice, const SimulationInfo *sim_info)
{
        AllSpikingSynapsesDeviceProperties allSynapses;
        BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * sim_info->totalNeurons;

        checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( sourceNeuronIndex, allSynapses.sourceNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( in_use, allSynapses.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
       
        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.count_neurons = 0;
}

/*
 *  Set some parameters used for advanceSynapsesDevice.
 */
void AllSpikingSynapses::setAdvanceSynapsesDeviceParams()
{
    setSynapseClassID();
}

/**
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllSpikingSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllSpikingSynapses;

    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses)) );
}

/*
 *  Advance all the Synapses in the simulation.
 *  Update the state of all synapses for a time step.
 *
 *  @param  allSynapsesDevice      Reference to the AllSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  allNeuronsDevice       Reference to the allNeurons struct on device memory.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 *  @param  sim_info               SimulationInfo class to read information from.
 */
void AllSpikingSynapses::advanceSynapses(void* allSynapsesDevice, void* allNeuronsDevice, void* synapseIndexMapDevice, const SimulationInfo *sim_info)
{
    if (total_synapse_counts == 0)
        return;

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( total_synapse_counts + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance synapses ------------->
    advanceSpikingSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( total_synapse_counts, (SynapseIndexMap*)synapseIndexMapDevice, g_simulationStep, sim_info->deltaT, (AllSpikingSynapsesDeviceProperties*)allSynapsesDevice );

    advanceSpikingSynapsesEventQueueDevice <<< 1, 1 >>> ((AllSpikingSynapsesDeviceProperties*)allSynapsesDevice);
}
